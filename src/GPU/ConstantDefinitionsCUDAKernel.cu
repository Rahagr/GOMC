
#include <hip/hip_runtime.h>
/*******************************************************************************
GPU OPTIMIZED MONTE CARLO (GOMC) 2.40
Copyright (C) 2018  GOMC Group
A copy of the GNU General Public License can be found in the COPYRIGHT.txt
along with this program, also can be found at <http://www.gnu.org/licenses/>.
********************************************************************************/
#ifdef GOMC_CUDA
#include <cuda.h>
#include <cuda_runtime.h>
#include "GeomLib.h"
#include "ConstantDefinitionsCUDAKernel.cuh"
#include <iostream>
#include <stdio.h>

void InitGPUForceField(VariablesCUDA &vars, real const *sigmaSq,
                       real const *epsilon_Cn,
                       real const *n, int VDW_Kind, int isMartini,
                       int count, real Rcut, real const *rCutCoulomb,
                       real RcutLow, real Ron, real const *alpha,
                       int ewald, real diElectric_1)
{
  int countSq = count * count;
  cudaMalloc(&vars.gpu_sigmaSq, countSq * sizeof(real));
  cudaMalloc(&vars.gpu_epsilon_Cn, countSq * sizeof(real));
  cudaMalloc(&vars.gpu_n, countSq * sizeof(real));
  cudaMalloc(&vars.gpu_VDW_Kind, sizeof(int));
  cudaMalloc(&vars.gpu_isMartini, sizeof(int));
  cudaMalloc(&vars.gpu_count, sizeof(int));
  cudaMalloc(&vars.gpu_rCut, sizeof(real));
  cudaMalloc(&vars.gpu_rCutCoulomb, BOX_TOTAL * sizeof(real));
  cudaMalloc(&vars.gpu_rCutLow, sizeof(real));
  cudaMalloc(&vars.gpu_rOn, sizeof(real));
  cudaMalloc(&vars.gpu_alpha, BOX_TOTAL * sizeof(real));
  cudaMalloc(&vars.gpu_ewald, sizeof(int));
  cudaMalloc(&vars.gpu_diElectric_1, sizeof(real));

  cudaMemcpy(vars.gpu_sigmaSq, sigmaSq, countSq * sizeof(real),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars.gpu_epsilon_Cn, epsilon_Cn, countSq * sizeof(real),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars.gpu_n, n, countSq * sizeof(real), cudaMemcpyHostToDevice);
  cudaMemcpy(vars.gpu_VDW_Kind, &VDW_Kind, sizeof(int),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars.gpu_isMartini, &isMartini, sizeof(int),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars.gpu_count, &count, sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(vars.gpu_rCut, &Rcut, sizeof(real), cudaMemcpyHostToDevice);
  cudaMemcpy(vars.gpu_rCutCoulomb, rCutCoulomb, BOX_TOTAL * sizeof(real),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars.gpu_rCutLow, &RcutLow, sizeof(real),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars.gpu_rOn, &Ron, sizeof(real), cudaMemcpyHostToDevice);
  cudaMemcpy(vars.gpu_alpha, alpha, BOX_TOTAL * sizeof(real),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars.gpu_ewald, &ewald, sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(vars.gpu_diElectric_1, &diElectric_1, sizeof(real),
             cudaMemcpyHostToDevice);
}

void InitCoordinatesCUDA(VariablesCUDA *vars, uint atomNumber,
                         uint maxAtomsInMol, uint maxMolNumber)
{
  cudaMalloc(&vars->gpu_x, atomNumber * sizeof(real));
  cudaMalloc(&vars->gpu_y, atomNumber * sizeof(real));
  cudaMalloc(&vars->gpu_z, atomNumber * sizeof(real));

  cudaMalloc(&vars->gpu_dx, atomNumber * sizeof(real));
  cudaMalloc(&vars->gpu_dy, atomNumber * sizeof(real));
  cudaMalloc(&vars->gpu_dz, atomNumber * sizeof(real));

  cudaMalloc(&vars->gpu_nx, maxAtomsInMol * sizeof(real));
  cudaMalloc(&vars->gpu_ny, maxAtomsInMol * sizeof(real));
  cudaMalloc(&vars->gpu_nz, maxAtomsInMol * sizeof(real));

  cudaMalloc(&vars->gpu_comx, maxMolNumber * sizeof(real));
  cudaMalloc(&vars->gpu_comy, maxMolNumber * sizeof(real));
  cudaMalloc(&vars->gpu_comz, maxMolNumber * sizeof(real));

  cudaMalloc(&vars->gpu_rT11, MAX_PAIR_SIZE * sizeof(real));
  cudaMalloc(&vars->gpu_rT12, MAX_PAIR_SIZE * sizeof(real));
  cudaMalloc(&vars->gpu_rT13, MAX_PAIR_SIZE * sizeof(real));
  cudaMalloc(&vars->gpu_rT22, MAX_PAIR_SIZE * sizeof(real));
  cudaMalloc(&vars->gpu_rT23, MAX_PAIR_SIZE * sizeof(real));
  cudaMalloc(&vars->gpu_rT33, MAX_PAIR_SIZE * sizeof(real));
  cudaMalloc(&vars->gpu_vT11, MAX_PAIR_SIZE * sizeof(real));
  cudaMalloc(&vars->gpu_vT12, MAX_PAIR_SIZE * sizeof(real));
  cudaMalloc(&vars->gpu_vT13, MAX_PAIR_SIZE * sizeof(real));
  cudaMalloc(&vars->gpu_vT22, MAX_PAIR_SIZE * sizeof(real));
  cudaMalloc(&vars->gpu_vT23, MAX_PAIR_SIZE * sizeof(real));
  cudaMalloc(&vars->gpu_vT33, MAX_PAIR_SIZE * sizeof(real));

  cudaMalloc(&vars->gpu_nonOrth, sizeof(int));
  vars->gpu_cell_x = new real *[BOX_TOTAL];
  vars->gpu_cell_y = new real *[BOX_TOTAL];
  vars->gpu_cell_z = new real *[BOX_TOTAL];
  vars->gpu_Invcell_x = new real *[BOX_TOTAL];
  vars->gpu_Invcell_y = new real *[BOX_TOTAL];
  vars->gpu_Invcell_z = new real *[BOX_TOTAL];
  for(uint b = 0; b < BOX_TOTAL; b++) {
    cudaMalloc(&vars->gpu_cell_x[b], 3 * sizeof(real));
    cudaMalloc(&vars->gpu_cell_y[b], 3 * sizeof(real));
    cudaMalloc(&vars->gpu_cell_z[b], 3 * sizeof(real));
    cudaMalloc(&vars->gpu_Invcell_x[b], 3 * sizeof(real));
    cudaMalloc(&vars->gpu_Invcell_y[b], 3 * sizeof(real));
    cudaMalloc(&vars->gpu_Invcell_z[b], 3 * sizeof(real));
  }
}

void InitEwaldVariablesCUDA(VariablesCUDA *vars, uint imageTotal)
{
  vars->gpu_kx = new real *[BOX_TOTAL];
  vars->gpu_ky = new real *[BOX_TOTAL];
  vars->gpu_kz = new real *[BOX_TOTAL];
  vars->gpu_kxRef = new real *[BOX_TOTAL];
  vars->gpu_kyRef = new real *[BOX_TOTAL];
  vars->gpu_kzRef = new real *[BOX_TOTAL];
  vars->gpu_sumRnew = new real *[BOX_TOTAL];
  vars->gpu_sumRref = new real *[BOX_TOTAL];
  vars->gpu_sumInew = new real *[BOX_TOTAL];
  vars->gpu_sumIref = new real *[BOX_TOTAL];
  vars->gpu_prefact = new real *[BOX_TOTAL];
  vars->gpu_prefactRef = new real *[BOX_TOTAL];
  vars->gpu_hsqr = new real *[BOX_TOTAL];
  vars->gpu_hsqrRef = new real *[BOX_TOTAL];

  for(uint b = 0; b < BOX_TOTAL; b++) {
    cudaMalloc(&vars->gpu_kx[b], imageTotal * sizeof(real));
    cudaMalloc(&vars->gpu_ky[b], imageTotal * sizeof(real));
    cudaMalloc(&vars->gpu_kz[b], imageTotal * sizeof(real));
    cudaMalloc(&vars->gpu_kxRef[b], imageTotal * sizeof(real));
    cudaMalloc(&vars->gpu_kyRef[b], imageTotal * sizeof(real));
    cudaMalloc(&vars->gpu_kzRef[b], imageTotal * sizeof(real));
    cudaMalloc(&vars->gpu_sumRnew[b], imageTotal * sizeof(real));
    cudaMalloc(&vars->gpu_sumRref[b], imageTotal * sizeof(real));
    cudaMalloc(&vars->gpu_sumInew[b], imageTotal * sizeof(real));
    cudaMalloc(&vars->gpu_sumIref[b], imageTotal * sizeof(real));

    cudaMalloc(&vars->gpu_prefact[b], imageTotal * sizeof(real));
    cudaMalloc(&vars->gpu_prefactRef[b], imageTotal * sizeof(real));
    cudaMalloc(&vars->gpu_hsqr[b], imageTotal * sizeof(real));
    cudaMalloc(&vars->gpu_hsqrRef[b], imageTotal * sizeof(real));
  }
}

void CopyCurrentToRefCUDA(VariablesCUDA *vars, uint box, uint imageTotal)
{
  cudaMemcpy(vars->gpu_sumRref[box], vars->gpu_sumRnew[box],
             imageTotal * sizeof(real), cudaMemcpyDeviceToDevice);
  cudaMemcpy(vars->gpu_sumIref[box], vars->gpu_sumInew[box],
             imageTotal * sizeof(real), cudaMemcpyDeviceToDevice);
  cudaMemcpy(vars->gpu_prefactRef[box], vars->gpu_prefact[box],
             imageTotal * sizeof(real), cudaMemcpyDeviceToDevice);
  cudaMemcpy(vars->gpu_hsqrRef[box], vars->gpu_hsqr[box],
             imageTotal * sizeof(real), cudaMemcpyDeviceToDevice);
  cudaMemcpy(vars->gpu_kxRef[box], vars->gpu_kx[box],
             imageTotal * sizeof(real), cudaMemcpyDeviceToDevice);
  cudaMemcpy(vars->gpu_kyRef[box], vars->gpu_ky[box],
             imageTotal * sizeof(real), cudaMemcpyDeviceToDevice);
  cudaMemcpy(vars->gpu_kzRef[box], vars->gpu_kz[box],
             imageTotal * sizeof(real), cudaMemcpyDeviceToDevice);
}

void UpdateRecipVecCUDA(VariablesCUDA *vars, uint box)
{
  real *tempKx, *tempKy, *tempKz, *tempHsqr, *tempPrefact;
  tempKx = vars->gpu_kxRef[box];
  tempKy = vars->gpu_kyRef[box];
  tempKz = vars->gpu_kzRef[box];
  tempHsqr = vars->gpu_hsqrRef[box];
  tempPrefact = vars->gpu_prefactRef[box];

  vars->gpu_kxRef[box] = vars->gpu_kx[box];
  vars->gpu_kyRef[box] = vars->gpu_ky[box];
  vars->gpu_kzRef[box] = vars->gpu_kz[box];
  vars->gpu_hsqrRef[box] = vars->gpu_hsqr[box];
  vars->gpu_prefactRef[box] = vars->gpu_prefact[box];

  vars->gpu_kx[box] = tempKx;
  vars->gpu_ky[box] = tempKy;
  vars->gpu_kz[box] = tempKz;
  vars->gpu_hsqr[box] = tempHsqr;
  vars->gpu_prefact[box] = tempPrefact;
}

void UpdateRecipCUDA(VariablesCUDA *vars, uint box)
{
  real *tempR, *tempI;
  tempR = vars->gpu_sumRref[box];
  tempI = vars->gpu_sumIref[box];
  vars->gpu_sumRref[box] = vars->gpu_sumRnew[box];
  vars->gpu_sumIref[box] = vars->gpu_sumInew[box];
  vars->gpu_sumRnew[box] = tempR;
  vars->gpu_sumInew[box] = tempI;
}

void UpdateCellBasisCUDA(VariablesCUDA *vars, uint box, real *cellBasis_x,
                         real *cellBasis_y, real *cellBasis_z)
{
  int nonOrth = 0;
  cudaMemcpy(vars->gpu_cell_x[box], cellBasis_x, 3 * sizeof(real),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_cell_y[box], cellBasis_y, 3 * sizeof(real),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_cell_z[box], cellBasis_z, 3 * sizeof(real),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_nonOrth, &nonOrth, sizeof(int), cudaMemcpyHostToDevice);
}

void UpdateInvCellBasisCUDA(VariablesCUDA *vars, uint box,
                            real *invCellBasis_x, real *invCellBasis_y,
                            real *invCellBasis_z)
{
  int nonOrth = 1;
  cudaMemcpy(vars->gpu_Invcell_x[box], invCellBasis_x, 3 * sizeof(real),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_Invcell_y[box], invCellBasis_y, 3 * sizeof(real),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_Invcell_z[box], invCellBasis_z, 3 * sizeof(real),
             cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_nonOrth, &nonOrth, sizeof(int), cudaMemcpyHostToDevice);
}

void DestroyEwaldCUDAVars(VariablesCUDA *vars)
{
  for(uint b = 0; b < BOX_TOTAL; b++) {
    cudaFree(vars->gpu_kx[b]);
    cudaFree(vars->gpu_ky[b]);
    cudaFree(vars->gpu_kz[b]);
    cudaFree(vars->gpu_kxRef[b]);
    cudaFree(vars->gpu_kyRef[b]);
    cudaFree(vars->gpu_kzRef[b]);
    cudaFree(vars->gpu_sumRnew[b]);
    cudaFree(vars->gpu_sumRref[b]);
    cudaFree(vars->gpu_sumInew[b]);
    cudaFree(vars->gpu_sumIref[b]);
    cudaFree(vars->gpu_prefact[b]);
    cudaFree(vars->gpu_prefactRef[b]);
    cudaFree(vars->gpu_hsqr[b]);
    cudaFree(vars->gpu_hsqrRef[b]);
  }
  delete [] vars->gpu_kx;
  delete [] vars->gpu_ky;
  delete [] vars->gpu_kz;
  delete [] vars->gpu_kxRef;
  delete [] vars->gpu_kyRef;
  delete [] vars->gpu_kzRef;
  delete [] vars->gpu_sumRnew;
  delete [] vars->gpu_sumRref;
  delete [] vars->gpu_sumInew;
  delete [] vars->gpu_sumIref;
  delete [] vars->gpu_prefact;
  delete [] vars->gpu_prefactRef;
  delete [] vars->gpu_hsqr;
  delete [] vars->gpu_hsqrRef;
}

void DestroyCUDAVars(VariablesCUDA *vars)
{
  cudaFree(vars->gpu_sigmaSq);
  cudaFree(vars->gpu_epsilon_Cn);
  cudaFree(vars->gpu_n);
  cudaFree(vars->gpu_VDW_Kind);
  cudaFree(vars->gpu_isMartini);
  cudaFree(vars->gpu_count);
  cudaFree(vars->gpu_rCut);
  cudaFree(vars->gpu_rCutCoulomb);
  cudaFree(vars->gpu_rCutLow);
  cudaFree(vars->gpu_rOn);
  cudaFree(vars->gpu_alpha);
  cudaFree(vars->gpu_ewald);
  cudaFree(vars->gpu_diElectric_1);
  cudaFree(vars->gpu_x);
  cudaFree(vars->gpu_y);
  cudaFree(vars->gpu_z);
  cudaFree(vars->gpu_dx);
  cudaFree(vars->gpu_dy);
  cudaFree(vars->gpu_dz);
  cudaFree(vars->gpu_nx);
  cudaFree(vars->gpu_ny);
  cudaFree(vars->gpu_nz);
  cudaFree(vars->gpu_comx);
  cudaFree(vars->gpu_comy);
  cudaFree(vars->gpu_comz);
  cudaFree(vars->gpu_rT11);
  cudaFree(vars->gpu_rT12);
  cudaFree(vars->gpu_rT13);
  cudaFree(vars->gpu_rT22);
  cudaFree(vars->gpu_rT23);
  cudaFree(vars->gpu_rT33);
  cudaFree(vars->gpu_vT11);
  cudaFree(vars->gpu_vT12);
  cudaFree(vars->gpu_vT13);
  cudaFree(vars->gpu_vT22);
  cudaFree(vars->gpu_vT23);
  cudaFree(vars->gpu_vT33);
  cudaFree(vars->gpu_nonOrth);
  for(uint b = 0; b < BOX_TOTAL; b++) {
    cudaFree(vars->gpu_cell_x[b]);
    cudaFree(vars->gpu_cell_y[b]);
    cudaFree(vars->gpu_cell_z[b]);
    cudaFree(vars->gpu_Invcell_x[b]);
    cudaFree(vars->gpu_Invcell_y[b]);
    cudaFree(vars->gpu_Invcell_z[b]);
  }
  delete [] vars-> gpu_cell_x;
  delete [] vars-> gpu_cell_y;
  delete [] vars-> gpu_cell_z;
  delete [] vars-> gpu_Invcell_x;
  delete [] vars-> gpu_Invcell_y;
  delete [] vars-> gpu_Invcell_z;
}

#endif /*GOMC_CUDA*/
